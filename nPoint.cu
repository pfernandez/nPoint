#include "hip/hip_runtime.h"
/*	Simulation overview:

	1. Define the information contained within a point in space as a c++ struct.
	2. Generate multiple copies of this point in a GPU array, each with unique parameters.
	3. Calculate a quantity at each point due to the given parameters within a CUDA kernel.
	4. Advance in time, and repeat.
*/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>



int numPoints = 4096,		// number of points to generate (multiples of 32 are best)
    maxSteps  = 250000, 	// total iterations
	saveSteps = 1000;		// data will be written to file when current step % saveSteps = 0


__device__ __constant__ float dt = 0.0005,   // Time step size.
                              A, k, w, c, dk;

const float pi = 3.14159265358979f;                          
const float plotBoundary = 4 * pi;


typedef struct
{
	// Represents a point in space with its associated fields, as well
	// as any information we wish to carry along with each point.
	
	float x, y,     // position data
		  psi;      // wave height
		  
} point;



// HOST FUNCTIONS //////////////////////////////////////////////////////////////////////////////////
//


// Get GPU debugging mesagges.
inline void check_cuda_errors( const char *filename, const int line_number )
{
#ifdef DEBUG
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if( error != hipSuccess )
  {
    printf( "CUDA error at %s:%i: %s\n", filename, line_number, hipGetErrorString( error ) );
    exit( -1 );
  }
#endif
}

// Get the current timestamp.
char *date_and_time()
{
	time_t  time_raw_format;
	struct tm * ptr_time;
	time ( &time_raw_format);
	ptr_time = localtime ( &time_raw_format );
	return asctime(ptr_time);
}

// Store simulation data into the directory specified by *path.
void writeDataToFile( char *path, point *p, int numPoints, float simTime, float plotLimit )
{
	// Initialize data file and write common parameters:
	FILE *file;	
	char filename[ strlen( path )+24 ];
	sprintf( filename,"%s/%.16f.dat", path, simTime );
	file = fopen( filename,"w" );
	char sp[10] = "         ";
	fprintf( file,"# limit = %g\n#\n# x%s  y%s  psi\n", plotLimit, sp, sp);
	
	// Write data to file:
	for( int i = 0; i < numPoints; i++ )
	{
		fprintf( file,"% .4e % .4e % .4e\n",
			p[i].x, p[i].y, p[i].psi );
	}
	
	fclose(file);
}


// Calculate constants used in device kernels.
void initializeDeviceConstants()
{
    float a = 1,
          lambda = 2 * pi,
          K = 2 * pi / lambda,
          hBar = 1 / ( 2 * pi ),  // letting h = 1
          m = 1,
          W = 0.5 * hBar * K * K / m,
          C =  W / K,
          DK = K / 8;  // determines the "tightness" of the wave packet

    hipMemcpyToSymbol(HIP_SYMBOL( A), &a, sizeof( float ) );
    hipMemcpyToSymbol(HIP_SYMBOL( k), &K, sizeof( float ) );
    hipMemcpyToSymbol(HIP_SYMBOL( w), &W, sizeof( float ) );
    hipMemcpyToSymbol(HIP_SYMBOL( c), &C, sizeof( float ) );
    hipMemcpyToSymbol(HIP_SYMBOL( dk), &DK, sizeof( float ) );
}


// Populate the point array on the host.
void initializePoints( point *p, int N )
{
    // Define the boudaries of the point-space.
    float dR = plotBoundary / sqrt( (float)N ),
          dTheta = 2 * pi / sqrt( (float)N );
          
    // Assign each point a specific location in space.
    int i = 0;
	for( float r = 0; r <= plotBoundary; r += dR )
	{	
	    for( float theta = 0; theta <= 2 * pi; theta += dTheta )
	    {
	        if( i > N )
	            continue;
            p[i].x = r * cos( theta );     // position vector
            p[i].y = r * sin( theta );	   //
            i++;
        }
	}
}

//
// END HOST FUNTIONS ///////////////////////////////////////////////////////////////////////////////




// DEVICE KERNELS //////////////////////////////////////////////////////////////////////////////////
//


__device__ int getGlobalIndex( void ) { return blockIdx.x * blockDim.x + threadIdx.x; }

__device__ float t = 0;
                              
__device__ void updateTime() { t += dt; }

// Calcutate the new point values in time and space at each timestep.
__global__ void incrementPoint( point *p, int numPoints )
{
	int	i = getGlobalIndex();
	 
    float r = sqrt( p[i].x*p[i].x + p[i].y*p[i].y );
    
    //p[i].psi = A * cos( k * r - w * t );  // simple circular wave
    
    // Disbursive Wave Packet
    // From A. C. Philips: Intro to Quantum Mechanics, problem 2.2 and fig. 2.1, with c = w/k.
    float alpha = r - c * t;
    p[i].psi = 2 * A * dk * sin( dk * alpha ) * cos( k * alpha ) / ( dk * alpha );
    
    updateTime();
}


//
// END DEVICE KERNELS //////////////////////////////////////////////////////////////////////////////


int main(int argc, char *argv[])
{


////////////////////////////////////////////////////////////////////////////////////////////////////
// SELECT DEVICE


    // Get the number of CUDA enabled devices.
	int deviceCount;
	hipGetDeviceCount( &deviceCount );
	if( deviceCount < 1 )
	{
		printf( "No CUDA-capable devices were detected.\n" );
		return 1;
	}

	// Get properties, select device, and display info.
	int device, useDevice;
	float computeCapability, cudaVersion = 0;
	hipDeviceProp_t deviceProp;
	for( device = 0; device < deviceCount; device++ )
	{
	    // Display info for each device.
		hipGetDeviceProperties( &deviceProp, device );
		computeCapability = deviceProp.major + deviceProp.minor * 0.1;
			
	    // Select the device with the greatest compute capability. If multiple devices
	    // with this capability are found, the first encountered will be used.
		if( computeCapability > cudaVersion )
		{
			cudaVersion = computeCapability;
			useDevice = device;
		}
	}
	device = useDevice;
	hipGetDevice(&device);
	hipGetDeviceProperties( &deviceProp, device );
    printf( "# Using %s, device number: %d, compute capability: %g.\n",
        deviceProp.name, device, cudaVersion );

    // Check for kernel timeout.
	if( deviceProp.kernelExecTimeoutEnabled == 1 )
	{
	    printf( "#\n" );
		printf( "# Warning: Timeout enabled. Using this device without a display \n" );
		printf( "# may lift kernel timeout restrictions.\n" );
	}  
	
	
// END SELECT DEVICE
////////////////////////////////////////////////////////////////////////////////////////////////////


	// Allocate memory:
	int threadsPerBlock = ( deviceProp.major >= 2 ? 512 : 256 ),
		numBlocks		= ceil( (float)numPoints / (float)threadsPerBlock ),
		numBytes		= numPoints * sizeof( point );
	point *cpuPointArray, *gpuPointArray;
	cpuPointArray = (point*)malloc( numBytes );
	hipMalloc( (void**)&gpuPointArray, numBytes );
	check_cuda_errors( __FILE__, __LINE__ );
	
	
////////////////////////////////////////////////////////////////////////////////////////////////////
// BEGIN SIMULATION


	// Begin simulation:
	for( int s = 0; s < maxSteps; s++ )
	{
		
		// Perform setup tasks before the first iteration.
		if( s == 0 )
		{	
		    // Initialize point array and device constants.
			srand( time( NULL ) );
			initializePoints( cpuPointArray, numPoints );
			initializeDeviceConstants();
			hipMemcpy( gpuPointArray, cpuPointArray, numBytes, hipMemcpyHostToDevice );

			// Write simulation info to stdout.
			float stepSize;
			hipMemcpyFromSymbol( &stepSize, HIP_SYMBOL(dt), sizeof(float) );
			printf( "#\n#\n" );
			printf( "# Started %s#\n# numPoints = %d\n# maxSteps  = %d\n# dt        = %g\n#\n",
				date_and_time(), numPoints, maxSteps, stepSize );
			printf( "# Steps   | Seconds  | Sim. Time\n# ------------------------------\n" );
			
			check_cuda_errors(__FILE__, __LINE__);	
		}

		// Execute portion of simulation on GPU:
		incrementPoint<<<numBlocks,threadsPerBlock>>>( gpuPointArray, numPoints );
			
		// Retrieve results:
		float simTime;
		hipMemcpyFromSymbol( &simTime, HIP_SYMBOL(t), sizeof( float ) );
		hipMemcpy( cpuPointArray, gpuPointArray, numBytes, hipMemcpyDeviceToHost );

		// Output data every [saveSteps] steps.
		if( s % saveSteps == 0 )
		{
		    // Write to file if save paths specified in argument to main().
		    if( argc > 1 )
			    writeDataToFile( argv[1], cpuPointArray, numPoints, simTime, plotBoundary );
			
			// Print data to stdout.
			printf( "%.4e %.4e %.4e\n", (float)s, (float)( clock() / CLOCKS_PER_SEC ), simTime );
		}

	}
	
	
// END SIMULATION
////////////////////////////////////////////////////////////////////////////////////////////////////
 	
 	
  	// Deallocate memory and exit.
  	free( cpuPointArray );
  	hipFree( gpuPointArray );
  	check_cuda_errors( __FILE__, __LINE__ );
  	printf( "\n# Completed %s", date_and_time() );
	return 0;
	
	
}	// end main
